#include "optimus/utils/cuda_utils.h"

namespace optimus {

template <typename T>
void check(T err, const char *const func, const char *const file,
           const int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

template void check<hipError_t>(hipError_t err, const char *const func,
                               const char *const file, const int line);

void checkLast(const char *const file, const int line) {
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

}  // namespace optimus
