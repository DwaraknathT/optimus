#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hiprand.h>
#include <iostream>
#include "optimus/kernels/ops/gemm.h"
#include "optimus/kernels/ops/affine_transform.h"
#include "optimus/utils/memanager.h"
#include "optimus/layers/dense.h"
#include "optimus/tensor.h"

using namespace optimus;

void GPU_fill_rand(float* A, int nr_rows_A, int nr_cols_A) {
    // Create a pseudo-random number generator
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
    // Set the seed for the random number generator using the system clock
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());
    // Fill the array with random numbers on the device
    hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

void gpu_blas_mmul(const float* A, const float* B, float* C, const int m,
                   const int k, const int n) {
    int lda = m, ldb = k, ldc = m;
    const float alf = 1;
    const float bet = 0;
    const float* alpha = &alf;
    const float* beta = &bet;
    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    // Do the actual multiplication
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B,
                ldb, beta, C, ldc);
    // Destroy the handle
    hipblasDestroy(handle);
}

void test_matmul() {
    const uint32_t m = 32 * 2048;
    const uint32_t n = 1024;
    const uint32_t k = 1024 * 3;
    const size_t size_a = sizeof(float) * m * n;
    const size_t size_b = sizeof(float) * n * k;
    const size_t size_c = sizeof(float) * m * k;
    const float alpha = 1.0;
    const float beta = 0;

    auto memory_manager = new optimus::MemManager();
    float* h_c =
        (float*)(memory_manager->allocate(size_c, optimus::MEMORY_CPU));

    float* d_a =
        (float*)(memory_manager->allocate(size_a, optimus::MEMORY_GPU));
    float* d_b =
        (float*)(memory_manager->allocate(size_b, optimus::MEMORY_GPU));
    float* d_c =
        (float*)(memory_manager->allocate(size_c, optimus::MEMORY_GPU));
        
    GPU_fill_rand(d_a, m, n);
    GPU_fill_rand(d_b, n, k);

    optimus::ops::InvokeGeMM<float>(d_a, d_b, d_c, m, n, k, alpha, beta);
    gpu_blas_mmul(d_a, d_b, d_c, m, n, k);

    hipMemcpy(d_c, h_c, size_c, hipMemcpyDeviceToHost);
    delete memory_manager;
}

void test_affine_transform() {
    const uint32_t m = 32 * 2048;
    const uint32_t n = 1024;
    const uint32_t k = 1024 * 3;
    const size_t size_a = sizeof(float) * m * n;
    const size_t size_b = sizeof(float) * n * k;
    const size_t size_bias = sizeof(float) * k;
    const size_t size_c = sizeof(float) * m * k;

    auto memory_manager = new optimus::MemManager();
    float* h_c =
        (float*)(memory_manager->allocate(size_c, optimus::MEMORY_CPU));

    float* d_a =
        (float*)(memory_manager->allocate(size_a, optimus::MEMORY_GPU));
    float* d_b =
        (float*)(memory_manager->allocate(size_b, optimus::MEMORY_GPU));
    float* d_bias =
        (float*)(memory_manager->allocate(size_bias, optimus::MEMORY_GPU));
    float* d_c =
        (float*)(memory_manager->allocate(size_c, optimus::MEMORY_GPU));

    GPU_fill_rand(d_a, m, n);
    GPU_fill_rand(d_b, n, k);
    GPU_fill_rand(d_bias, 1, k);

    optimus::ops::InvokeAffineTransformation<float>(d_a, d_b, d_bias, d_c, m, n, k);

    hipMemcpy(d_c, h_c, size_c, hipMemcpyDeviceToHost);
    delete memory_manager;
}

int main() {
    auto a = optimus::Tensor<int>({32, 1024, 512});
    auto stride = a.stride();
    for (int i : stride) {
        std::cout << i << " ";
    }
    // auto layer = new optimus::layers::Dense<float>(32, 64, optimus::MEMORY_GPU);
    // delete layer;
    // test_affine_transform();
    return 0;
}
