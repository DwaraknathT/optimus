#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include "hip/hip_runtime.h"
#include "optimus/tensor.h"
#include "optimus/utils/array_utils.h"
#include "optimus/utils/cuda_utils.h"

using namespace optimus;

__global__ void setkernel(int* data, int* stride) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    int k = blockIdx.x;
    int val = i * 100 + j * 10 + k;
    data[getOffset({i, j, k}, stride)] = val;
}

__global__ void myKernel(int* data, int* stride) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    int k = blockIdx.x;
    printf("(%d, %d, %d) -> %d\n", i, j, k, data[getOffset({i, j, k}, stride)]);
}

int main() {

    int shape[] = {2, 3, 4};
    auto arr = new optimus::Tensor<int>({2, 3, 4}, optimus::MEMORY_GPU);

    dim3 grid(shape[2], 1);
    dim3 block(shape[0], shape[1]);
    setkernel<<<grid, block>>>(arr->data, arr->stride_);
    myKernel<<<grid, block>>>(arr->data, arr->stride_);
    hipDeviceSynchronize();
    CHECK_LAST_CUDA_ERROR();
    
    return 0;
}
